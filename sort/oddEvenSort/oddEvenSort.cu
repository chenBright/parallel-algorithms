#include <iostream>
 using namespace std;
 
 #include <hip/hip_runtime.h>
 
 __device__ void swap(int *a, int *b) {
     int temp = *a;
     *a = *b;
     *b = temp;
 }
 
 __global__ void sort(int *d_arr, int n, bool isEven) {
     int i;
     if (isEven) {
         i = threadIdx.x * 2;
     } else {
         i = threadIdx.x * 2 + 1;
     }
 
     if (i < n -1) {
         if (d_arr[i] > d_arr[i + 1]) {
             swap(&d_arr[i], &d_arr[i + 1]);
         }
     }
 }
 
 void sort(int *arr, int n) {
     size_t size = n * sizeof(int);
 
     int *d_arr = NULL;
     hipMalloc((void **)&d_arr, size);
     hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);
 
 
     for (int phase = 0; phase < n; ++phase) {
         sort<<<1, n / 2>>>(d_arr, n, phase % 2 == 0);
     }
 
     hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);
 
 }
 
 int main() {
     int h_a[] = {9, 8, 7, 6, 5, 4, 3, 2, 1, 0};
     int n = 10;
 
     for (int i = 0; i < n; ++i) {
         cout << h_a[i] << ' ';
     }
     cout << endl;
 
 
     sort(h_a, n);
 
     for (int i = 0; i < n; ++i) {
         cout << h_a[i] << ' ';
     }
     cout << endl;
 
     return 0;
 }
 
 